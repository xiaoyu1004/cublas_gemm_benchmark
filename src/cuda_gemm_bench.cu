#include <chrono>
#include <cstdint>
#include <iomanip>
#include <iostream>
#include <memory>
#include <sstream>
#include <stdexcept>
#include <tuple>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "tensor.h"

#ifndef PAD_KERNELS
#define PAD_KERNELS 1
#endif

// Vector saves m, n, k, a_t, b_t
std::vector<std::tuple<int, int, int, bool, bool>> inference_server_set = {
    std::make_tuple(5124, 700, 2048, false, false),
    std::make_tuple(35, 700, 2048, false, false),
    std::make_tuple(5124, 700, 2560, false, false),
    std::make_tuple(35, 700, 2560, false, false),
    std::make_tuple(5124, 1500, 2048, false, false),
    std::make_tuple(35, 1500, 2048, false, false),
    std::make_tuple(5124, 1500, 2560, false, false),
    std::make_tuple(35, 1500, 2560, false, false),
    std::make_tuple(7680, 1, 2560, false, false),
    std::make_tuple(7680, 2, 2560, false, false),
    std::make_tuple(7680, 4, 2560, false, false),
    std::make_tuple(3072, 1, 1024, false, false),
    std::make_tuple(3072, 2, 1024, false, false),
    std::make_tuple(3072, 4, 1024, false, false),
    std::make_tuple(512, 1, 500000, false, false),
    std::make_tuple(1024, 1, 500000, false, false),
    std::make_tuple(512, 2, 500000, false, false),
    std::make_tuple(1024, 2, 500000, false, false),
    std::make_tuple(512, 4, 500000, false, false),
    std::make_tuple(1024, 4, 500000, false, false),
    std::make_tuple(1024, 700, 512, false, false),
    std::make_tuple(7680, 1500, 2560, false, false),
    std::make_tuple(6144, 1500, 2048, false, false),
    std::make_tuple(4608, 1500, 1536, false, false),
    std::make_tuple(8448, 1500, 2816, false, false),
    std::make_tuple(3072, 1500, 1024, false, false),
    std::make_tuple(7680, 3000, 2560, false, false),
    std::make_tuple(6144, 3000, 2048, false, false),
    std::make_tuple(4608, 3000, 1536, false, false),
    std::make_tuple(8448, 3000, 2816, false, false),
    std::make_tuple(3072, 3000, 1024, false, false),
    std::make_tuple(7680, 6000, 2560, false, false),
    std::make_tuple(6144, 6000, 2048, false, false),
    std::make_tuple(4608, 6000, 1536, false, false),
    std::make_tuple(8448, 6000, 2816, false, false),
    std::make_tuple(3072, 6000, 1024, false, false),
    std::make_tuple(6144, 1, 2048, false, false),
    std::make_tuple(4608, 1, 1536, false, false),
    std::make_tuple(8448, 1, 2816, false, false),
    std::make_tuple(6144, 2, 2048, false, false),
    std::make_tuple(4608, 2, 1536, false, false),
    std::make_tuple(8448, 2, 2816, false, false),
    std::make_tuple(6144, 4, 2048, false, false),
    std::make_tuple(4608, 4, 1536, false, false),
    std::make_tuple(8448, 4, 2816, false, false),
    std::make_tuple(512, 1500, 2816, false, false),
    std::make_tuple(512, 1500, 2048, false, false),
    std::make_tuple(512, 1500, 2560, false, false),
    std::make_tuple(512, 1500, 1536, false, false),
    std::make_tuple(1024, 1500, 2816, false, false),
    std::make_tuple(1024, 1500, 2048, false, false),
    std::make_tuple(1024, 1500, 2560, false, false),
    std::make_tuple(1024, 1500, 1536, false, false),
    std::make_tuple(512, 1, 512, false, false),
    std::make_tuple(1024, 1, 512, false, false),
    std::make_tuple(512, 3000, 2816, false, false),
    std::make_tuple(512, 3000, 2048, false, false),
    std::make_tuple(512, 3000, 2560, false, false),
    std::make_tuple(512, 3000, 1536, false, false),
    std::make_tuple(1024, 3000, 2816, false, false),
    std::make_tuple(1024, 3000, 2048, false, false),
    std::make_tuple(1024, 3000, 2560, false, false),
    std::make_tuple(1024, 3000, 1536, false, false),
    std::make_tuple(512, 2, 512, false, false),
    std::make_tuple(1024, 2, 512, false, false),
    std::make_tuple(512, 6000, 2816, false, false),
    std::make_tuple(512, 6000, 2048, false, false),
    std::make_tuple(512, 6000, 2560, false, false),
    std::make_tuple(512, 6000, 1536, false, false),
    std::make_tuple(1024, 6000, 2816, false, false),
    std::make_tuple(1024, 6000, 2048, false, false),
    std::make_tuple(1024, 6000, 2560, false, false),
    std::make_tuple(1024, 6000, 1536, false, false),
    std::make_tuple(512, 4, 512, false, false),
    std::make_tuple(1024, 4, 512, false, false)};

/*
Usage:

The default precision is set based on the architecture and mode.

By default, the program runs the benchmark in training mode.

bin/gemm_bench

To run inference mode, use the following command:

bin/gemm_bench inference


To change the precision for training/inference, use:

bin/gemm_bench train <precision>
bin/gemm_bench inference <precision>

Supported precision types:

For Maxwell GPUS:
float for training and inference

For Pascal GPUS:
float, half for training
float, half, int8 for inference

*/

template <typename T1, typename T2>
int time_gemm(Tensor<T1> A, Tensor<T1> B, Tensor<T2> C, bool a_t, bool b_t,
              hipblasHandle_t cublas_handle, bool use_tensor_core)
{
    const int alpha = 1.f;
    const int beta = 1.f;

    int m = C.dims()[0];
    int k = a_t ? A.dims()[0] : A.dims()[1];
    int n = C.dims()[1];

    int numRepeats = 6;
    hipblasStatus_t stat;

    hipDataType A_type = HIP_R_32F;
    hipDataType B_type = HIP_R_32F;
    hipDataType C_type = HIP_R_32F;
    hipDataType compute_type = HIP_R_32F;
    hipblasGemmAlgo_t algo;

    if (std::is_same<T1, uint16_t>::value)
    {
        A_type = HIP_R_16F;
        B_type = HIP_R_16F;
        C_type = HIP_R_16F;
        compute_type = HIP_R_16F;
    }

    if (std::is_same<T1, uint8_t>::value)
    {
        A_type = HIP_R_8I;
        B_type = HIP_R_8I;
        C_type = HIP_R_32I;
        compute_type = HIP_R_32I;
    }

    algo = use_tensor_core ? CUBLAS_GEMM_DFALT_TENSOR_OP : HIPBLAS_GEMM_DEFAULT;

    stat =
        hipblasGemmEx(cublas_handle, a_t ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                     b_t ? HIPBLAS_OP_T : HIPBLAS_OP_N, m, n, k, &alpha, A.begin(),
                     A_type, A.dims()[0], B.begin(), B_type, B.dims()[0], &beta,
                     C.begin(), C_type, C.dims()[0], compute_type, algo);

    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        throw std::runtime_error("sgemm failed");
    }

    hipDeviceSynchronize();

    auto start = std::chrono::steady_clock::now();

    for (int i = 0; i < numRepeats; ++i)
    {
        stat = hipblasGemmEx(cublas_handle, a_t ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                            b_t ? HIPBLAS_OP_T : HIPBLAS_OP_N, m, n, k, &alpha,
                            A.begin(), A_type, A.dims()[0], B.begin(), B_type,
                            B.dims()[0], &beta, C.begin(), C_type, C.dims()[0],
                            compute_type, algo);

        if (stat != HIPBLAS_STATUS_SUCCESS)
        {
            throw std::runtime_error("sgemm failed");
        }
    }
    hipDeviceSynchronize();

    auto end = std::chrono::steady_clock::now();

    return static_cast<int>(
        std::chrono::duration<double, std::micro>(end - start).count() /
        numRepeats);
}

int main(int argc, char **argv)
{
    // Get Device Number
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n",
               static_cast<int>(error_id), hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }

    int inference = 1;
    if (argc > 1)
    {
        std::string inf = "inference";
        inference = argv[1] == inf ? 1 : 0;
    }

    if (inference)
    {
        std::cout << "Running inference benchmark " << std::endl;
    }
    else
    {
        std::cout << "Running training benchmark " << std::endl;
    }

    for (int dev = 0; dev < deviceCount; ++dev)
    {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout << "Device " << dev << ": " << deviceProp.name << std::endl;

        hiprandGenerator_t curand_gen;
        hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
        hiprandSetPseudoRandomGeneratorSeed(curand_gen, 123ULL);

        hipblasHandle_t cublas_handle;
        hipblasStatus_t status = hipblasCreate(&cublas_handle);
        if (status != HIPBLAS_STATUS_SUCCESS)
        {
            std::cout << "CUBLAS init failed" << std::endl;
        }

        std::cout << "m\tn\tk\ta_t\tb_t\tfp32 perf\tfp16 perf\tint8 perf\tfp16 tensor core perf\tint8 tensor core perf" << std::endl;

        int pad_kernels_count = 0;

        for (const auto &problem : inference_server_set)
        {
            int m, n, k;
            bool a_t, b_t;
            std::tie(m, n, k, a_t, b_t) = problem;
            int time_ms;

            double flops = (double)m * n * k * 2;
            double perf = 0.f;

            std::cout << m << "\t";
            std::cout << n << "\t";
            std::cout << k << "\t";
            std::cout << "n" << "\t";
            std::cout << "n";

            // set cublas to not use tensor core
            status = hipblasSetMathMode(cublas_handle, HIPBLAS_DEFAULT_MATH);
            if (status != HIPBLAS_STATUS_SUCCESS)
            {
                std::cout << "CUBLAS math mode failed" << std::endl;
            }

            // fp32 benchmark
            {
                auto a = rand<float>({a_t ? k : m, a_t ? m : k}, curand_gen);
                auto b = rand<float>({b_t ? n : k, b_t ? k : n}, curand_gen);
                auto c = zeros<float>({m, n});
                time_ms = time_gemm<float, float>(a, b, c, a_t, b_t, cublas_handle, false);
                perf = flops / (time_ms * 1000.0);
                std::cout << "  " << std::setprecision(6) << perf;
            }

            // fp16 benchmark
            {
                auto a = rand<uint16_t>({a_t ? k : m, a_t ? m : k}, curand_gen);
                auto b = rand<uint16_t>({b_t ? n : k, b_t ? k : n}, curand_gen);
                auto c = zeros<uint16_t>({m, n});
                time_ms = time_gemm<uint16_t, uint16_t>(a, b, c, a_t, b_t, cublas_handle, false);
                perf = flops / (time_ms * 1000.0);
                std::cout << "  " << std::setprecision(6) << perf;
            }

            // int8 benchmark
            {
                int pad_m;
                pad_m = m;
                if (pad_m % 4)
                {
                    pad_kernels_count++;
                    pad_dim(pad_m, 4);
                }

                auto a = rand<uint8_t>({a_t ? k : pad_m, a_t ? pad_m : k}, curand_gen);
                auto b = rand<uint8_t>({b_t ? n : k, b_t ? k : n}, curand_gen);
                auto c = zeros<int>({pad_m, n});
                time_ms = time_gemm<uint8_t, int>(a, b, c, a_t, b_t, cublas_handle, false);
                perf = flops / (time_ms * 1000.0);
                std::cout << "  " << std::setprecision(6) << perf;
            }

            // set cublas to use tensor core
            status = hipblasSetMathMode(cublas_handle, HIPBLAS_TENSOR_OP_MATH);
            if (status != HIPBLAS_STATUS_SUCCESS)
            {
                std::cout << "CUBLAS math mode failed" << std::endl;
            }

            // fp16 tensor core benchmark
            {
                auto a = rand<uint16_t>({a_t ? k : m, a_t ? m : k}, curand_gen);
                auto b = rand<uint16_t>({b_t ? n : k, b_t ? k : n}, curand_gen);
                auto c = zeros<uint16_t>({m, n});
                time_ms = time_gemm<uint16_t, uint16_t>(a, b, c, a_t, b_t, cublas_handle, true);
                perf = flops / (time_ms * 1000.0);
                std::cout << "  " << std::setprecision(6) << perf;
            }

            // int8 tensor core benchmark
            {
                int pad_m;
                pad_m = m;
                if (pad_m % 4)
                {
                    pad_kernels_count++;
                    pad_dim(pad_m, 4);
                }

                auto a = rand<uint8_t>({a_t ? k : pad_m, a_t ? pad_m : k}, curand_gen);
                auto b = rand<uint8_t>({b_t ? n : k, b_t ? k : n}, curand_gen);
                auto c = zeros<int>({pad_m, n});
                time_ms = time_gemm<uint8_t, int>(a, b, c, a_t, b_t, cublas_handle, true);
                perf = flops / (time_ms * 1000.0);
                std::cout << "  " << std::setprecision(6) << perf;
            }

            // std::stringstream ss;
            // ss << "Unsupported precision requested. Precision: " << precision << "
            // Inference: " << inference;

            std::cout << std::endl;
        }

        hipblasDestroy(cublas_handle);
        hiprandDestroyGenerator(curand_gen);
    }

    return 0;
}
